#include "hip/hip_runtime.h"
/**
 * @File		main.cu
 * @brief		
 * @author		Jeong Hoon (Sian) Choi
 * @version 	1.0.0
 * @date		2024-05-19
 */

/* Copyright (C)
 * 2024 - Jeong Hoon (Sian) Choi
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <utility>
#include <algorithm>
#include <numeric>

#include <string_view>
#include <string>
#include <vector>

#include <thread>
#include <mutex>
#include <shared_mutex>
#include <condition_variable>

#include "sian/timer.h"

#if _TARGET_OS == OS_WINDOWS

#elif _TARGET_OS == OS_LINUX

// void custom_terminate_fnct(void) {
//	exit(1);
// }

template <typename T>
bool check_matrix(const T* a, const T* b, const int n, const int m, double tolerance = 1e-5) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			int index = i * m + j;
			if (std::abs(a[index] - b[index]) > tolerance) return false;
		}
	}
	return true;
}

template <typename T>
void single_thread(const T* a, const T* b, T* c, const int n, const int m, const int k) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			int inner = 0;
			for (int l = 0; l < k; ++l) {
				inner += a[i * k + l] + b[l * m + j];
			}
			c[i * k + j] = inner;
		}
	}
}

template <typename T>
void multi_thread(const T* a, const T* b, T* c, const int n, const int m, const int k,
				  const int thread_index, const int thread_num) {
	int tasks = std::ceil(static_cast<float>(m) / thread_num);
	for (int i = 0; i < n; ++i) {
		for (int j = thread_index * tasks; j < (thread_index + 1) * tasks; ++j) {
			int value = 0;
			for (int l = 0; l < k; ++l) {
				if (j > m) break;
				value += a[i * k +l] + b[l * m + j];
			}
			c[i * k + j] = value;
		}
	}
}

static const int block_size = 32;

template <typename T>
__global__ void cuda_kernel(const T* a, const T* b, T* c, const int n, const int m, const int k) {
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int local_col = threadIdx.x;
	int local_row = threadIdx.y;
	
	__shared__ T partial_a[block_size][block_size];
	__shared__ T partial_b[block_size][block_size];

	for (int blk = 0; blk < std::ceil(static_cast<float>(k) / block_size); ++blk) {
		int value = 0;
		int stride = blk * block_size;

	    if (row >= m || stride + local_row >= k)
			partial_a[local_row][local_col] = 0;
		else
			partial_a[local_row][local_col] = a[row * k + (stride + local_col)];

		if (col >= n || stride + local_col >= k)
			partial_b[local_row][local_col] = 0;
		else
			partial_b[local_row][local_col] = b[(stride + local_row) * m + col];
			
		__syncthreads();

		for (int i = 0; i < block_size; ++i) {
			value += partial_a[local_row][local_col] * partial_b[local_row][local_col];
		}

		__syncthreads();

		if (row < m && col > n)
			c[m * row + col] += value;
	}
}

int main(int argc, char* argv[]) {
//	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	sian::Timer timer(3);
	
	const int n = 2048;
	const int k = 2048;
	const int m = 2048;

	double* a = new double[n * k];
	double* b = new double[k * m];
	double* c1 = new double[n * m];
	double* c2 = new double[n * m];
	double* c3 = new double[n * m];

	std::cout << "####\nMatrix Multiply Parallel Calculation\n####\n" << std::endl;
	
	timer[0].set_name("single thread");
	timer[0].start();
	single_thread(a, b, c1, n, m ,k);
	timer[0].stop();
	
	timer[1].set_name("multi threads");
	const auto thread_num = std::thread::hardware_concurrency();
	std::vector<std::thread> threads;
	timer[1].start();
	for (int i = 0; i < thread_num; ++i) {
		threads.emplace_back(&multi_thread<double>, a, b, c2, n, m, k, i, thread_num);
	}

	for (auto& thread : threads) thread.join();
	timer[1].stop();
	std::cout << "multi thread is correct : " << std::boolalpha << check_matrix(c1, c2, n, m) << std::endl;

	timer[2].set_name("cuda GPU SIMT");
	timer[2].start();

	dim3 grid_dim(std::ceil(static_cast<float>(n) / block_size),
				  std::ceil(static_cast<float>(m) / block_size));
	dim3 block_dim(block_size, block_size);
	cuda_kernel<double><<<grid_dim, block_dim>>>(a, b, c2, n, m ,k);
	hipDeviceSynchronize();
	timer[2].stop();
	std::cout << "cuda GPU SIMT is correct : " << std::boolalpha << check_matrix(c1, c2, n, m) << std::endl;
	
	std::cout << timer;

	delete[] c2;
	delete[] c1;
	delete[] b;
	delete[] a;

	return 0;
}

#endif // OS dependency
