#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define NUM_STREAMS	4

__globa__ void kernel(int* in, int* out) {
	// workload;
}

int main(void) {
	hipStream_t stream[NUM_STREAMS];
	hipEvent_t start[NUM_STREAMS], end[NUM_STREAMS];

	for(int i=0; i<NUM_STREAMS; ++i) {
		hipStreamCreate(&stream[i]);
		hipEventCreate(&start[i]);
		hipEventCreate(&end[i]);
	}
	//
	// omit
	//
	int chunkSize=ARRAY_SIZE/NUM_STREAMS;

	int offset[NUM_STREAMS]={0, };
	for(int i=0; i<NUM_STREAMS; ++i) offset[i]=chunkSize*i;

	for(int i=0; i<NUM_STREAMS; ++i) {
		hipEventRecord(start[i], stream[i]);
		hipMemcpyAsync(dIn+offset[i], in+offset[i], sizeof(int)*chunkSize, hipMemcpyHostToDevice, stream[i]);
	}

	for(int i=0; i<NUM_STREAMS; ++i) {
		kernel<<<chunkSize/1024, 1024, 0, stream[i]>>>(dIn+offset[i], dOut+offset[i]);
	}

	for(int i=0; i<NUM_STREAMS; ++i) {
		hipMemcpyAsync(out+offset[i], dOut+offset[i], sizeof(int)*chunkSize, hipMemcpyDeviceToHost, stream[i]);
		hipEventRecord(end[i], stream[i]);
	}

	hipDeviceSynchronize();
	
	for(int i=0; i<NUM_STREAMS; ++i) {
		float time=0;
		hipEventElapsedTime(&time, start[i], end[i]);
		printf("Stream[%d] : %f ms\n", i, time);
	}

	return 0;
}