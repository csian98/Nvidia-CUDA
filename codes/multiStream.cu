#include <hip/hip_runtime.h>



#define NUM_BLOCK		(128*1024)
#define ARRAY_SIZE	(1024*NUM_BLOCK)
#define NUM_STREAMS	4
#define WORK_LOAD		256

__global__ void kernel(int* _in, int* _out) {
	int tID=blockIdx.x*blockDim.x+threadIdx.x;

	int temp=0;
	int in=_in[tID];

	for(int i=0; i<WORK_LOAD; ++i) {
		temp=(temp+in*5)%10;
	}
	_out[tID]=temp;
}

int main(void) {
	int *in=NULL, *out=NULL, *dIn=NULL, *dOut=NULL;

	hipHostMalloc(&in, sizeof(int)*ARRAY_SIZE, hipHostMallocDefault);
	hipHostMalloc(&out, sizeof(int)*ARRAY_SIZE, hipHostMallocDefault);
	memset(in, 0, sizeof(int)*ARRAY_SIZE);
	memset(out, 0, sizeof(int)*ARRAY_SIZE);

	hipMalloc(&dIn, sizeof(int)*ARRAY_SIZE);
	hipMalloc(&dOut, sizeof(int)*ARRAY_SIZE);
	for(int i=0; i<ARRAY_SIZE; ++i) in[i]=rand()%10;

	hipStream_t stream[NUM_STREAMS];
	for(int i=0; i<NUM_STREAMS; ++i) hipStreamCreate(&stream[i]);

	int chunkSize=ARRAY_SIZE/NUM_STREAMS;

	for(int i=0; i<NUM_STREAMS; ++i) {
		int offset=chunkSize*i;
		hipMemcpyAsync(dIn+offset, in+offset, sizeof(int)*chunkSize, hipMemcpyHostToDevice, stream[i]);
	}

	for(int i=0; i<NUM_STREAMS; ++i) {
		int offset=chunkSize*i;
		kernel<<<NUM_BLOCK/NUM_STREAMS, 1024, 0, stream[i]>>>(dIn+offset, dOut+offset);
	}

	for(int i=0; i<NUM_STREAMS; ++i) {
		int offset=chunkSize*i;
		hipMemcpyAsync(out+offset, dOut+offset, sizeof(int)*chunkSize, hipMemcpyDeviceToHost, stream[i]);
	}

	hipDeviceSynchronize();

	for(int i=0; i<NUM_STREAMS; ++i) hipStreamDestroy(stream[i]);

	hipFree(dIn); hipFree(dOut);
	hipHostFree(in); hipHostFree(out);

	return 0;
}
