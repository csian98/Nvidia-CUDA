#include <hip/hip_runtime.h>



__global__ void kernel(int* in, int* out) {
	int deviceID;
	hipGetDevice(&deviceID);
	//
}

int main(void) {
	int ngpus;
	hipGetDeviceCount(&ngpus);

	for(int i=0; i<ngpus; ++i) {
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		//
	}

	for(int i=0; i<ngpus; ++i) {
		hipSetDevice(i);
		//cudaMemcpy(...);
		//kernel0<<<...>>>(...);
		//cudaMemcpy(...);
	}
}