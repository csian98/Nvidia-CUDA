#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

int main(void) {
	hipStream_t stream;
	hipStreamCreate(&stream);
	//
	Kernel<<<dimGrid, dimBlock, 0, stream>>>();

	hipStreamDestroy(stream);
}